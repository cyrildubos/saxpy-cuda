#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void initialize(float *x, float a, float *y, float b, size_t n)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
    {
        x[i] = a;
        y[i] = b;
    }
}

__global__ void saxpy(float a, float *x, float *y, size_t n)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += gridDim.x * blockDim.x)
        y[i] = a * x[i] + y[i];
}

int main()
{
    int device;
    hipGetDevice(&device);

    hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, device);

    const size_t n = 1 << 24;

    float *x;
    float *y;
    hipMallocManaged(&x, n * sizeof(float));
    hipMallocManaged(&y, n * sizeof(float));

    size_t grid = 32 * properties.multiProcessorCount;
    size_t block = 256;

    initialize<<<grid, block>>>(x, 2, y, 3, n);

    saxpy<<<grid, block>>>(5, x, y, n);

    hipDeviceSynchronize();

    hipMemPrefetchAsync(y, n, hipCpuDeviceId);

    printf("y[0] = %f\n", y[0]);
    printf("y[n - 1] = %f\n", y[n - 1]);

    return EXIT_SUCCESS;
}